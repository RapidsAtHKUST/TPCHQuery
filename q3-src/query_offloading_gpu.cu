#include "hip/hip_runtime.h"
#include <fstream>
#include <vector>

#include "util/thread_pool.h"
#include "index_query_helper.h"

#include "cuda/primitives.cuh"
#include "cuda/cuda_base.cuh"
#include "cuda/CUDAStat.cuh"

#include "file_loader_gpu.h"
#include "file_loader.h"

#define GetIndexArr GetMallocPReadArrReadOnlyGPU
#define ORDER_MAX_ID (600000010)

using namespace std;

IndexHelper::IndexHelper(string order_path, string line_item_path) {
    auto num_devices = 1;
    hipGetDeviceCount(&num_devices);
    log_info("Number of GPU devices: %d.", num_devices);

    order_keys_arr.resize(num_devices);
    item_order_keys_arr.resize(num_devices);
    item_prices_arr.resize(num_devices);
    bmp_arr.resize(num_devices);
    dict_arr.resize(num_devices);

    acc_prices_arr.resize(num_devices);
    acc_prices_filtered_arr.resize(num_devices);
    order_offset_arr.resize(num_devices);
    order_offset_filtered_arr.resize(num_devices);

#pragma omp parallel num_threads(num_devices)
    {
        auto gpu_id = omp_get_thread_num();
        hipSetDevice(gpu_id);
#ifdef UM
        CUDA_MALLOC(&acc_prices_arr[gpu_id], sizeof(double) * MAX_NUM_ORDERS, nullptr);
        CUDA_MALLOC(&acc_prices_filtered_arr[gpu_id], sizeof(double) * MAX_NUM_ORDERS, nullptr);
        CUDA_MALLOC(&order_offset_arr[gpu_id], sizeof(uint32_t) * MAX_NUM_ORDERS, nullptr);
        CUDA_MALLOC(&order_offset_filtered_arr[gpu_id], sizeof(uint32_t) * MAX_NUM_ORDERS, nullptr);
#else
        checkCudaErrors(hipMalloc((void**)&acc_prices_arr[gpu_id], sizeof(double) * MAX_NUM_ORDERS));
        checkCudaErrors(hipMalloc((void**)&acc_prices_filtered_arr[gpu_id], sizeof(double) * MAX_NUM_ORDERS));
        checkCudaErrors(hipMalloc((void**)&order_offset_arr[gpu_id], sizeof(uint32_t) * MAX_NUM_ORDERS));
        checkCudaErrors(hipMalloc((void**)&order_offset_filtered_arr[gpu_id], sizeof(uint32_t) * MAX_NUM_ORDERS));
#endif
    }

    // Load Order.
    string order_key_path = order_path + ORDER_KEY_BIN_FILE_SUFFIX;
    string order_date_path = order_path + ORDER_DATE_BIN_FILE_SUFFIX;
    string order_meta_path = order_path + ORDER_META_BIN_FILE_SUFFIX;
    {
        ifstream ifs(order_meta_path, std::ifstream::in);
        Archive<ifstream> ar(ifs);

        const char *test_chars = "BUILDING";
        ar >> category_table_ >> min_order_date_ >> max_order_date_
           >> order_second_level_range_ >> order_num_buckets_ >> order_bucket_ptrs_;
        log_info("Probe Test: %d", LinearProbe(category_table_, test_chars, 0, strlen(test_chars)));
    }
    for (auto &category: category_table_) {
        category.PrintStr();
    }
    size_of_orders_ = order_bucket_ptrs_.back();
    num_categories_ = category_table_.size();
    log_info("%d, %d, %d, %d, %d, %zu, %d", num_categories_, min_order_date_, max_order_date_,
             order_second_level_range_, order_num_buckets_, order_bucket_ptrs_.size(), size_of_orders_);
//    cout << order_bucket_ptrs_ << endl;
    int fd;

    ThreadPool pool(num_devices*2);
    vector<future<void>> futures;

    for(auto i = 0; i < num_devices; i++) {
        futures.emplace_back(pool.enqueue([=](){
            int fd;
            auto gpu_id = i;
            hipSetDevice(gpu_id);
            order_keys_arr[gpu_id] = GetIndexArr<int32_t>(order_key_path.c_str(), fd, size_of_orders_);

            auto &bmp = bmp_arr[gpu_id];
            auto &order_pos_dict = dict_arr[gpu_id];
#ifdef UM
            CUDA_MALLOC(&bmp, sizeof(bool) * ( ORDER_MAX_ID + 1), nullptr);
            CUDA_MALLOC(&order_pos_dict, sizeof(uint32_t) * (ORDER_MAX_ID + 1), nullptr);
#else
            checkCudaErrors(hipMalloc((void**)&bmp, sizeof(bool) * ( ORDER_MAX_ID + 1)));
            checkCudaErrors(hipMalloc((void**)&order_pos_dict, sizeof(uint32_t) * (ORDER_MAX_ID + 1)));
#endif
        }));
    }

    order_dates_ = GetMMAPArrReadOnly<uint32_t>(order_date_path.c_str(), fd, size_of_orders_);
    log_info("Finish Order Index Loading...Not Populate Yet");

    // Load LineItem.
    string item_order_id_path = line_item_path + LINE_ITEM_ORDER_KEY_FILE_SUFFIX;
    string item_price_path = line_item_path + LINE_ITEM_PRICE_FILE_SUFFIX;
    string item_meta_path = line_item_path + LINE_ITEM_META_BIN_FILE_SUFFIX;
    {
        ifstream ifs(item_meta_path, std::ifstream::in);
        Archive<ifstream> ar(ifs);
        ar >> min_ship_date_ >> max_ship_date_ >> item_num_buckets_ >> item_bucket_ptrs_;
    }
    size_of_items_ = item_bucket_ptrs_.back();
    log_info("%d, %d, %d, %zu, %d", min_ship_date_, max_ship_date_, item_num_buckets_, item_bucket_ptrs_.size(),
             size_of_items_);

    for(auto i = 0; i < num_devices; i++) {
        futures.emplace_back(pool.enqueue([=](){
            int fd = 0;
            auto gpu_id = i;
            hipSetDevice(gpu_id);
            item_order_keys_arr[gpu_id] = GetIndexArr<int32_t>(item_order_id_path.c_str(), fd, size_of_items_);
            item_prices_arr[gpu_id] = GetIndexArr<double>(item_price_path.c_str(), fd, size_of_items_);
        }));
    }
    for(auto &future: futures) {
        future.get();
    }

    log_info("Finish LineItem Loading...Not Populate Yet");
}

__global__
void buildBooleanArray(
        uint32_t start_pos, uint32_t end_pos,
        int32_t *order_keys_, bool *bmp, uint32_t *order_pos_dict) {
    auto gtid = threadIdx.x + blockDim.x * blockIdx.x + start_pos;
    auto gtnum = blockDim.x * gridDim.x;

    while (gtid < end_pos) {
        auto order_key = order_keys_[gtid];
        bmp[order_key] = true;
        order_pos_dict[order_key] = gtid - start_pos;
        gtid += gtnum;
    }
}

__global__
void filterJoin(
        uint32_t start_pos, uint32_t end_pos,
        int32_t *item_order_keys_, double *acc_prices, double *item_prices_, int32_t max_order_id,
        bool *bmp, uint32_t *order_pos_dict) {
    auto gtid = threadIdx.x + blockDim.x * blockIdx.x + start_pos;
    auto gtnum = blockDim.x * gridDim.x;

    while (gtid < end_pos) {
        auto order_key = item_order_keys_[gtid];
        if ((order_key <= max_order_id) && (bmp[order_key])) {
            atomicAdd(&acc_prices[order_pos_dict[order_key]], item_prices_[gtid]);
        }
        gtid += gtnum;
    }
}

void IndexHelper::evaluateWithGPU(
        vector<int32_t *> order_keys_arr, uint32_t order_bucket_ptr_beg, uint32_t order_bucket_ptr_end,
        vector<int32_t *> item_order_keys_arr, uint32_t lineitem_bucket_ptr_beg, uint32_t lineitem_bucket_ptr_end,
        vector<bool*> bmp_arr, vector<uint32_t *> dict_arr,
        vector<double *> item_prices_arr, uint32_t order_array_view_size, int lim, int32_t &size_of_results, Result *t) {
    CUDAMemStat memstat_detail;
    CUDATimeStat timing_detail;
    auto memstat = &memstat_detail;
    auto timing = &timing_detail;

    log_trace("Evaluate with GPUs");

    auto num_devices = 1;
    hipGetDeviceCount(&num_devices);
    log_info("Number of GPU devices: %d.", num_devices);

    auto lineitem_tuples_per_gpu = (lineitem_bucket_ptr_end - lineitem_bucket_ptr_beg + num_devices - 1) / num_devices;

    Timer timer;

    /*compute max_order_id with a single GPU*/
    hipSetDevice(0);
    int32_t max_order_id = CUBMax(&order_keys_arr[0][order_bucket_ptr_beg], (order_bucket_ptr_end - order_bucket_ptr_beg),
                                  memstat, timing);
    log_info("BMP Size: %d", max_order_id + 1);
    log_info("After get max_order_id: %.2f s.", timer.elapsed());

    hipDeviceSynchronize();

#pragma omp parallel num_threads(num_devices)
    {
        auto gpu_id = omp_get_thread_num();
        log_info("TID: %d, BMP Size: %d", gpu_id, max_order_id + 1);

        hipSetDevice(gpu_id);

        auto lineitem_bucket_ptr_beg_gpu = lineitem_bucket_ptr_beg + gpu_id * lineitem_tuples_per_gpu;
        auto lineitem_bucket_ptr_end_gpu = lineitem_bucket_ptr_beg + (gpu_id+1) * lineitem_tuples_per_gpu;
        if (lineitem_bucket_ptr_end_gpu > lineitem_bucket_ptr_end)
            lineitem_bucket_ptr_end_gpu = lineitem_bucket_ptr_end;
        log_info("GPU ID: %d, lineitem range: [%d, %d)", gpu_id, lineitem_bucket_ptr_beg_gpu, lineitem_bucket_ptr_end_gpu);

        checkCudaErrors(hipMemset(acc_prices_arr[gpu_id], 0, sizeof(double) * order_array_view_size));
        log_info("After malloc acc_prices_arr: %.2f s.", timer.elapsed());

        /*construct the mapping*/
        auto bmp = bmp_arr[gpu_id];
        auto order_pos_dict = dict_arr[gpu_id];
        checkCudaErrors(hipMemset(bmp, 0, sizeof(bool) * (max_order_id + 1)));

        log_info("TID: %d, Before Construction Data Structures: %.6lfs", gpu_id, timer.elapsed());

        /*build the boolean filter*/
        execKernel(buildBooleanArray, 1024, 256, timing, false,
                   order_bucket_ptr_beg, order_bucket_ptr_end,
                   order_keys_arr[gpu_id], bmp, order_pos_dict);

        log_info("TID: %d, Before Aggregation: %.6lfs", gpu_id,  timer.elapsed());

        execKernel(filterJoin, 1024, 256, timing, false,
                   lineitem_bucket_ptr_beg_gpu, lineitem_bucket_ptr_end_gpu,
                   item_order_keys_arr[gpu_id], acc_prices_arr[gpu_id], item_prices_arr[gpu_id], max_order_id, bmp, order_pos_dict);

        if (gpu_id != 0) //prefetch to gpu0
        {
#ifdef UM
            checkCudaErrors(hipMemPrefetchAsync(acc_prices_arr[gpu_id], sizeof(double)*order_array_view_size, 0));
#endif
        }
        log_info("TID: %d, Before Select: %.6lfs", gpu_id, timer.elapsed());
    }

    /*add up the acc_prices*/
    auto iter_begin = thrust::make_counting_iterator(0u);
    auto iter_end = thrust::make_counting_iterator(order_array_view_size);

    hipSetDevice(0);
    for(auto i = 1; i < num_devices; i++) {
        double *acc_prices_0 = acc_prices_arr[0];
        double *acc_prices_i = acc_prices_arr[i];
        timingKernel(
                thrust::transform(thrust::device, iter_begin, iter_end, acc_prices_arr[0], [=]
                __device__(uint32_t idx) {
                return acc_prices_0[idx] + acc_prices_i[idx];
        }), timing);
    }

    /*processing the summarized acc_prices*/
    bool *flag_is_zero = nullptr;
#ifdef UM
    CUDA_MALLOC(&flag_is_zero, sizeof(bool) * order_array_view_size, memstat);
#else
    checkCudaErrors(hipMalloc(&flag_is_zero, sizeof(bool) * order_array_view_size));
#endif

    /*the acc_price double buffer*/
    auto *acc_price_temp = acc_prices_filtered_arr[0];
    auto *acc_prices = acc_prices_arr[0];
    log_info("Before processing the summarized acc_prices: %.2f s.", timer.elapsed());

    /*the order_offset double buffer*/
    auto *order_offset = order_offset_arr[0];
    auto *order_offset_temp = order_offset_filtered_arr[0];

    thrust::counting_iterator<uint32_t> iter(order_bucket_ptr_beg);
    timingKernel(
            thrust::copy(iter, iter + order_array_view_size, order_offset), timing);

    /*construct the boolean filter*/
    timingKernel(
            thrust::transform(thrust::device, iter_begin, iter_end, flag_is_zero, [=]
            __device__(uint32_t idx) {
            return acc_prices[idx] > 0.0;
    }), timing);

    /*filter the acc_price*/
    size_of_results = CUBSelect(acc_prices, acc_price_temp, flag_is_zero, order_array_view_size, memstat, timing);
    CUBSelect(order_offset, order_offset_temp, flag_is_zero, order_array_view_size, memstat, timing);

    log_info("Non Zeros: %zu", size_of_results);

    /*CUB sort pairs*/
    void *d_temp_storage = nullptr;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes, acc_price_temp, acc_prices, order_offset_temp, order_offset, size_of_results);
#ifdef UM
    CUDA_MALLOC(&d_temp_storage, temp_storage_bytes, memstat);
#else
    checkCudaErrors(hipMalloc(&d_temp_storage, temp_storage_bytes));
#endif
    hipcub::DeviceRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes, acc_price_temp, acc_prices, order_offset_temp, order_offset, size_of_results);
    checkCudaErrors(hipDeviceSynchronize());

    for (auto i = 0; i < lim; i++) {
        t[i].price = acc_prices[i];
        t[i].order_offset = order_offset[i];
    }

    log_info("Maximal device memory demanded: %ld bytes.", memstat->get_max_use());
    log_info("Unfreed device memory size: %ld bytes.", memstat->get_cur_use());
}